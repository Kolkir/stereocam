#include "hip/hip_runtime.h"
#include "cu_median.h"
#include "cu_utils.h"

#include <cassert>

namespace cuda {

Median3DFilter::Median3DFilter(size_t width, size_t height, size_t blockSize, size_t depth)
    : width(width)
    , height(height)
    , depth(depth)
    , blockSize(blockSize)
    , frameBytesNumber(width * height){
    assert(depth >= 1);
    assert(width >= 1 && height >= 1);
    CHECK(hipMalloc(&frames, frameBytesNumber * depth));
    CHECK(hipMalloc(&cudaResult, frameBytesNumber));
    framesQueue.push(0);
}

Median3DFilter::~Median3DFilter()
{
    CHECK(hipFree(frames));
    CHECK(hipFree(cudaResult));
}

void Median3DFilter::pushFrame(const unsigned char *data)
{
    size_t frameIndex = 0;
    if (framesQueue.size() == depth){
        frameIndex = framesQueue.front();
        framesQueue.pop();
    } else {
        frameIndex = framesQueue.back() + 1;
    }

    CHECK(hipMemcpy(frames + (frameIndex * frameBytesNumber), data, frameBytesNumber, hipMemcpyHostToDevice));

    framesQueue.push(frameIndex);
}

__global__ void medianOnGPU(unsigned char* frames, unsigned char* result,
                            size_t width, size_t height, size_t depth,
                            int searchNum,
                            size_t frameSize) {
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int iz = threadIdx.z + blockIdx.z * blockDim.z;//blockIdx.z - should be always zero
    if (ix < width && iy < height && iz < depth) {
        unsigned int idx = iy * width + ix;
        iz *= frameSize;
        //search element position in sorted array
        int pos = 0;
        for (size_t i = 0; i < depth; ++i){
            if (i != iz && frames[iz + idx] > frames[i * frameSize + idx]){
                ++pos;
            }
        }

        if (pos == searchNum){
            result[idx] = frames[iz + idx];
        }
    }
}

void Median3DFilter::getFilteredFrame(unsigned char *data)
{
    dim3 block (blockSize, blockSize, depth);
    //std::cout << "Block size : " << block.x << " " << block.y << " " << block.z << std::endl;

    dim3 grid  ((width  + block.x - 1) / block.x,
                (height + block.y - 1) / block.y,
                (depth  + block.z - 1) / block.z);    
    //std::cout << "Grid size : " << grid.x << " " << grid.y << " " << grid.z << std::endl;

    medianOnGPU<<< grid, block >>>(frames, cudaResult, width, height, depth, depth / 2, frameBytesNumber);

    CHECK(hipPeekAtLastError());
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(data, cudaResult, frameBytesNumber, hipMemcpyDeviceToHost));
}

size_t Median3DFilter::getWidth() const {
    return width;
}

size_t Median3DFilter::getHeight() const {
    return height;
}

}
